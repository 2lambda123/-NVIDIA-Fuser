#include "hip/hip_runtime.h"
// clang-format off
/*
 * SPDX-FileCopyrightText: Copyright (c) 2023-present NVIDIA CORPORATION & AFFILIATES.
 * All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 */
// clang-format on
namespace warp {

template <typename T>
__device__ __forceinline__ T shfl_xor(T var, int laneMask, int width = 32) {
  return __shfl_xor_sync(0xffffffff, var, laneMask, width);
}
template <typename T>
__device__ __forceinline__ std::complex<T> shfl_xor(
    std::complex<T> var,
    int laneMask,
    int width = 32) {
  T real = __shfl_xor_sync(0xffffffff, var.real(), laneMask, width);
  T imag = __shfl_xor_sync(0xffffffff, var.imag(), laneMask, width);
  return std::complex<T>(real, imag);
}

template <typename T>
__device__ __forceinline__ T
warp_broadcast(T var, int srcLane, int width = 32) {
  return __shfl_sync(0xffffffff, var, srcLane, width);
}

template <typename T>
__device__ __forceinline__ std::complex<T> warp_broadcast(
    std::complex<T> var,
    int width = 32) {
  T real = __shfl_xor_sync(0xffffffff, var.real(), srcLane, width);
  T imag = __shfl_xor_sync(0xffffffff, var.imag(), srcLane, width);
  return std::complex<T>(real, imag);
}

template <
    bool SINGLE_WARP,
    bool Aligned,
    bool Padded,
    typename T,
    typename Func>
__device__ void warpReduceTIDX(
    T& out,
    const T& inp_val,
    Func reduction_op,
    T* shared_mem,
    bool read_write_pred,
    T init_val) {
  constexpr int WARP_SIZE = 32;

  T reduce_val = init_val;

  // Do warp reduction
  if (read_write_pred) {
    reduce_val = inp_val;
  }

  // Reduce within each warp
  // Register usage is reduced when Padded is true due to the elimination of the
  // if-statement.
  unsigned int warp_idx = threadIdx.x / WARP_SIZE;
  unsigned int lane_idx = threadIdx.x % WARP_SIZE;
  unsigned int reduction_size = blockDim.x;
  unsigned int num_of_warps = (reduction_size + WARP_SIZE - 1) / WARP_SIZE;
  int hoist_idx = reduction_size - lane_idx - warp_idx * WARP_SIZE;
  for (int i = 16; i >= 1; i /= 2) {
    T shuffled_value = shfl_xor(reduce_val, i, WARP_SIZE);
    if (Padded || i < hoist_idx) {
      reduction_op(reduce_val, shuffled_value);
    }
  }

  // Reduce across warp if needed
  // Load value to shared mem
  if (!SINGLE_WARP) {
    unsigned int reduce_group_id = threadIdx.z * blockDim.y + threadIdx.y;
    bool is_warp_head = lane_idx == 0;

    unsigned int smem_offset = reduce_group_id * num_of_warps;

    block_sync::sync<Aligned>();

    if (is_warp_head) {
      shared_mem[smem_offset + warp_idx] = reduce_val;
    }

    block_sync::sync<Aligned>();

    if (warp_idx == 0) {
      // This assumes num_of_warps will be < 32, meaning < 1024 threads.
      //  Should be true for long enough.
      assert(num_of_warps <= 32);

      reduce_val = lane_idx < num_of_warps ? shared_mem[smem_offset + lane_idx]
                                           : init_val;

      // Reduce within warp 0
      for (int i = 16; i >= 1; i /= 2) {
        reduction_op(reduce_val, shfl_xor(reduce_val, i, 32));
      }
    }

    if (is_warp_head) {
      reduction_op(out, reduce_val);
    }
    // needs sync, otherwise other warps may access shared memory before this
    // reduction is done.
    block_sync::sync<Aligned>();
  } else {
    if (!Padded) {
      reduce_val = warp_broadcast(reduce_val, 0);
    }
    reduction_op(out, reduce_val);
  }
}

} // namespace warp
