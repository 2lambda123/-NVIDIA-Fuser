#include "hip/hip_runtime.h"
// clang-format off
/*
 * SPDX-FileCopyrightText: Copyright (c) 2023-present NVIDIA CORPORATION & AFFILIATES.
 * All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 */
// Utility macro for this file
#define DEVICE_INLINE __device__ inline

// Utility for converting generic pointer to SMEM pointer in PTX.
//  We should review vectorized load/stores with shared memory.
//  SMEM memory movement PTX is only Global -> SMEM, SMEM -> Local, Local ->
//  SMEM, and this is needed for these PTX instructions to provide the SMEM
//  pointer.
DEVICE_INLINE unsigned toSmem(const void* raw_ptr) {
  unsigned smem_ptr_uint;
  asm("{ .reg .u64 smem_ptr; cvta.to.shared.u64 smem_ptr, %1; cvt.u32.u64 %0, smem_ptr; }\n"
      : "=r"(smem_ptr_uint)
      : "l"(raw_ptr));

  return smem_ptr_uint;
}

DEVICE_INLINE unsigned toSmem(unsigned addr) {
  // already converted
  return addr;
}

#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 750))

namespace Turing {

namespace util {

// LdMatrix has .x1, .x2 and .x4 options, currently we actively use .x2 and
//  .x4. In .x2 option. the the address register of upper half warp (lane 16-31)
//  are un-used but on Turing [sm75,sm80) architecture these un-used addresses
//  need to be valid, in the sense that:
//     1. The data it points to has to be within allocated shared mem buffer.
//     2. The address needs to be aligned to 16 byte.
//  See also:
// https://docs.nvidia.com/cuda/parallel-thread-execution/index.html#warp-level-matrix-instructions-ldmatrix
//  This function addresses 2. above by masking out the sub-16B component
//    of the address in upper warp and 1. is guaranteed by ldmatrix swizzle
//    util.
//  This will **not** affect any functionality. This is just modification
//    of unused pointers to satisfy the alignment requirement on Turing
//    hardware.
//  The alignment requirement is lifted on sm80+,
//    so this function is a no-op on Ampere or above.
DEVICE_INLINE void adjustPartialLdMatrixAddrInTuring(unsigned& addr_in_byte) {
#if (__CUDA_ARCH__ < 800)
  const unsigned thread_id = threadIdx.x;
  // Upper half warp has 8 bytes offset from aligned in .x2 option
  //  of ldmatrix. Currently no support for .x1 so assume always
  //  adjust by half warp.
  constexpr unsigned half_warp = 16;
  // Need to adjust to 16 byte alignment, mask out un-aligned component.
  constexpr unsigned mask_out = 16 - 1;
  // Adjust only in upper half warp.
  // use bit math to reduce strength
  if (thread_id & half_warp) {
    // mask out the bits where adjust_mask has 1.
    addr_in_byte &= (~mask_out);
  }
#endif //(__CUDA_ARCH__ < 800)
}

} // namespace util

// Load Matrix (per warp instruction) is to take data from SMEM to Local Memory.
//   Automatically handles vectorized loads/stores in the MMA operation.
//   Loads 8x8 matrix into a warp. Thread 0-7 provide the ptr that is the start
//   of each row. All other threads can simply point to something valid
//   (including 0).
// The x2 modifier on the instruction will actually load 2x8 rows to make a
// 16x8,
//   then thread 0-15 will specify the start of each row.
// Finally is an x4 modifier producing a 32x8 using addrs from 0-31 in each
// warp.
DEVICE_INLINE void ldMatrix(Array<__half, 4, 4>& out, unsigned addr) {
  uint2& val = reinterpret_cast<uint2&>(out);
  util::adjustPartialLdMatrixAddrInTuring(addr);
  asm volatile("ldmatrix.sync.aligned.x2.m8n8.shared.b16 {%0,%1}, [%2];"
               : "=r"(val.x), "=r"(val.y)
               : "r"(addr));
}

// Same as previous, 8x8 matrix is vectorized loaded, then scattered (to perform
// transpose) so threads will hold 2 values down a column (instead of the
// previous instruction that's across a row).
DEVICE_INLINE void ldMatrixT(Array<__half, 4, 4>& out, unsigned addr) {
  uint2& val = reinterpret_cast<uint2&>(out);
  util::adjustPartialLdMatrixAddrInTuring(addr);
  asm volatile("ldmatrix.sync.aligned.x2.trans.m8n8.shared.b16 {%0,%1}, [%2];"
               : "=r"(val.x), "=r"(val.y)
               : "r"(addr));
}

DEVICE_INLINE void ldMatrix(Array<__half, 8, 8>& out, unsigned addr) {
  uint4& val = reinterpret_cast<uint4&>(out);
  asm volatile("ldmatrix.sync.aligned.x4.m8n8.shared.b16 {%0,%1,%2,%3}, [%4];"
               : "=r"(val.x), "=r"(val.y), "=r"(val.z), "=r"(val.w)
               : "r"(addr));
}

DEVICE_INLINE void ldMatrixT(Array<__half, 8, 8>& out, unsigned addr) {
  uint4& val = reinterpret_cast<uint4&>(out);
  asm volatile(
      "ldmatrix.sync.aligned.x4.trans.m8n8.shared.b16 {%0,%1,%2,%3}, [%4];"
      : "=r"(val.x), "=r"(val.y), "=r"(val.z), "=r"(val.w)
      : "r"(addr));
}

} // namespace Turing

#endif // Arch 75

#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 800))

namespace Ampere {

// MMA instruction wrappers (sm_80+):
template<int dst_size, typename word_type>
DEVICE_INLINE void cpGlobalToShared(
        unsigned smem_addr,
        const void *gmem_ptr,
        int src_size) {
    constexpr int word_size = sizeof(word_type);
    const auto *__restrict src = reinterpret_cast<const word_type *>(__builtin_assume_aligned(gmem_ptr, word_size));
    auto *__restrict dst = reinterpret_cast<word_type *>(__builtin_assume_aligned(__cvta_shared_to_generic(smem_addr), word_size));
    // these allow to generate ld.global.u8 and st.shared.u8 properly.
    bool s = __isShared(dst);
    __builtin_assume(s);
    bool g = __isGlobal(src);
    __builtin_assume(g);

    bool b = src_size >= 0 && src_size < dst_size;
    __builtin_assume(b);

#pragma unroll
    for (int i = 0; i < dst_size; ++i) {
        dst[i] = i < src_size ? src[i] : word_type{0};
    }
}

/**
 * Copies src_size bytes from gmem_ptr to smem_addr. 
 * Zero fills upper bytes if src_size < dst_size.
 * @tparam dst_size 
 * @param smem_addr 
 * @param gmem_ptr 
 * @param src_size_or_predicate 
 */
template<int dst_size, typename Predicate_T>
DEVICE_INLINE void cpAsyncCaAligned(
        unsigned smem_addr,
        const void *gmem_ptr,
        Predicate_T src_size_or_predicate) {
    static_assert(dst_size == 4 || dst_size == 8 || dst_size == 16, "cp_async : unsupported byte size");

    static_assert(is_same_v < Predicate_T, bool > || is_same_v < Predicate_T, int > );

    if constexpr(is_same_v < Predicate_T, bool > )
    {
        asm volatile(
                "{\n"
                "  .reg .pred p;\n"
                "  setp.eq.b32 p, %3, 0;\n"
                "cp.async.ca.shared.global [%0], [%1], %2, p;\n"
                "}\n"::"r"(smem_addr),
        "l"(__builtin_assume_aligned(gmem_ptr, dst_size)),
        "n"(dst_size),
        "r"((int) src_size_or_predicate) : "memory");
    } else if constexpr(is_same_v < Predicate_T, int > )
    {
        asm volatile(
                "{\n"
                "cp.async.ca.shared.global [%0], [%1], %2, %3;\n"
                "}\n"::"r"(smem_addr),
        "l"(__builtin_assume_aligned(gmem_ptr, dst_size)),
        "n"(dst_size),
        "r"(src_size_or_predicate): "memory");
    }
}


/**
 * Async load from gmem to smem. 
 * @tparam T elt type
 * @tparam count elt count
 * @tparam assume_aligned_data Assumes that gmem is aligned on sizeof(T) * count
 * @param smem_addr 
 * @param gmem_ptr must be at least aligned on sizeof(T)
 * @param src_size_or_predicate if predicate is false, zeroes are written to smem and gmem is not accessed
 */
template<typename Predicate_T, typename T, int count, bool assume_aligned_data = true>
DEVICE_INLINE void cpAsyncCa(
        unsigned smem_addr,
        const T *gmem_ptr,
        Predicate_T src_size_or_predicate) {

    static_assert(is_same_v < Predicate_T, bool > || is_same_v < Predicate_T, int > );

    constexpr unsigned byte_size = sizeof(T) * count;
    static_assert(byte_size == 4 || byte_size == 8 || byte_size == 16, "cp_async : unsupported byte size");

    if constexpr(assume_aligned_data || sizeof(T) == byte_size)
    {
        const char *__restrict src = reinterpret_cast< const char *>(__builtin_assume_aligned(gmem_ptr, byte_size));
        cpAsyncCaAligned<byte_size, Predicate_T>(smem_addr, src, src_size_or_predicate);
    } else {
        const auto ptr_low_bytes = static_cast<unsigned> (reinterpret_cast<size_t>(gmem_ptr) & 0xFFFFFFFF); //Should be std::intptr_t
        if (ptr_low_bytes % byte_size == 0) {
            // ptr is aligned on cp size, 
            const char *__restrict src = reinterpret_cast< const char *>(__builtin_assume_aligned(gmem_ptr, byte_size));
            cpAsyncCaAligned<byte_size, Predicate_T>(smem_addr, src, src_size_or_predicate);
        } else { // Unaligned case, fallback on min_alignment
            constexpr unsigned min_alignment = sizeof(T); // min alignment guaranteed for the type
            static_assert(byte_size % min_alignment == 0);
            if constexpr(min_alignment == 1)
            { // fallback to copy through registers using bytes
                const char *__restrict src = reinterpret_cast< const char *>(__builtin_assume_aligned(gmem_ptr, 1));
                cpGlobalToShared<byte_size, unsigned char>(smem_addr, src, src_size_or_predicate);
            } else if constexpr(min_alignment == 2)
            { // fallback to copy through registers using shorts
                const char *__restrict src = reinterpret_cast< const char *>(__builtin_assume_aligned(gmem_ptr, 2));
                cpGlobalToShared<byte_size / 2, unsigned short>(smem_addr, src, src_size_or_predicate);
            } else {
                static_assert(min_alignment == 4 || min_alignment == 8 || min_alignment == 16, "cp_async : unsupported elt size");
                const char *__restrict src = reinterpret_cast< const char *>(__builtin_assume_aligned(gmem_ptr, min_alignment));
#pragma unroll
                for (unsigned i = 0; i < byte_size; i += min_alignment) {
                    cpAsyncCaAligned<min_alignment>(smem_addr + i, src + i, src_size_or_predicate);
                }
            }
        }
    }
}


template<typename Predicate_T>
DEVICE_INLINE void cpAsyncCgAligned(
        unsigned smem_addr,
        void const *__restrict gmem_ptr,
        Predicate_T src_size_or_predicate) {
    static_assert(is_same_v < Predicate_T, bool > || is_same_v < Predicate_T, int > );
    if constexpr(is_same_v < Predicate_T, bool > )
    {
        asm volatile(
                "{\n"
                "  .reg .pred p;\n"
                "  setp.eq.b32 p, %2, 0;\n"
                "cp.async.cg.shared.global [%0], [%1], 16, p;\n"
                "}\n"::"r"(smem_addr),
        "l"(__builtin_assume_aligned(gmem_ptr, 16)),
        "r"((int) src_size_or_predicate) : "memory");
    } else if constexpr(is_same_v < Predicate_T, int > )
    {
        asm volatile(
                "{\n"
                "cp.async.cg.shared.global [%0], [%1], 16, %2;\n"
                "}\n"::"r"(smem_addr),
        "l"(__builtin_assume_aligned(gmem_ptr, 16)),
        "r"(src_size_or_predicate) : "memory");
    }
}


// Global to SMEM load that is asynchronous,
// not guaranteed to be completed until cpAsyncBarrier() is called.
template<typename Predicate_T, typename T, int len, bool assume_aligned_data = true>
DEVICE_INLINE void cpAsyncCg(
        unsigned smem_addr,
        const void *gmem_ptr,
        Predicate_T src_size_or_predicate) {
    constexpr int byte_size = sizeof(T) * len;
    static_assert(byte_size == 16, "cp_async : unsupported byte size");

    if constexpr(assume_aligned_data || sizeof(T) == 16)
    {
        cpAsyncCgAligned(smem_addr, gmem_ptr, src_size_or_predicate);
    } else {
        const auto ptr_low = static_cast<unsigned> (reinterpret_cast<size_t>(gmem_ptr) & 0xFFFFFFFF); //Should be std::intptr_t
        unsigned misalignment = ptr_low % byte_size;
        __builtin_assume(misalignment < byte_size);
        if (misalignment == 0) {
            cpAsyncCgAligned(smem_addr, gmem_ptr, src_size_or_predicate);
        } else {
            cpAsyncCa<Predicate_T, T, len, assume_aligned_data>(smem_addr, gmem_ptr, src_size_or_predicate);
        }
    }
}

// TODO: Might have a different category of sync if we want to build out this:
DEVICE_INLINE void cpAsyncBarrier() {
  asm volatile("cp.async.wait_all;");
}

DEVICE_INLINE void cpAsyncCommit() {
  asm volatile("cp.async.commit_group;");
}

template <int keep_stages>
DEVICE_INLINE void cpAsyncPartialBarrier() {
  asm volatile("cp.async.wait_group %0;\n" ::"n"(keep_stages));
}

} // namespace Ampere

#endif // Arch 80

// Double buffer calculation utilities:

// In place update of double buffer index that has been accumulated to the data
// buffer.
template <int number_of_stage, int loop_offset>
DEVICE_INLINE void doubleBufferUpdate(
    DataPointer& data_buffer,
    const nvfuser_index_t& loop_index,
    nvfuser_index_t buffer_size) {
  // static_assert(
  //     loop_offset < number_of_stage && loop_offset > -number_of_stage);

  // convert offset to [0, number_of_stage)
  constexpr nvfuser_index_t offset =
      loop_offset < 0 ? (loop_offset + number_of_stage) : loop_offset;

  // Rewind back at number_of_stage-1, otherwise increment by 1.
  nvfuser_index_t increment =
      (loop_index % number_of_stage) == (number_of_stage - 1 - offset)
      ? buffer_size * (-number_of_stage + 1)
      : buffer_size;
  data_buffer += increment;
}

template <int number_of_stage, int loop_offset>
DEVICE_INLINE void doubleBufferUpdate(
    unsigned& data_buffer,
    const nvfuser_index_t& loop_index,
    nvfuser_index_t buffer_size) {
  // static_assert(
  //     loop_offset < number_of_stage && loop_offset > -number_of_stage);

  // convert offset to [0, number_of_stage)
  constexpr nvfuser_index_t offset =
      loop_offset < 0 ? (loop_offset + number_of_stage) : loop_offset;

  // Rewind back at number_of_stage-1, otherwise increment by 1.
  nvfuser_index_t increment =
      (loop_index % number_of_stage) == (number_of_stage - 1 - offset)
      ? buffer_size * (-number_of_stage + 1)
      : buffer_size;
  data_buffer += (unsigned)increment;
}

// Update double buffer offset value for smem double buffered tensors.
// See [Uniform Double Buffer Offset]
template <int number_of_stage, int loop_offset>
DEVICE_INLINE void doubleBufferSwitch(
    int& buffer_offset,
    const nvfuser_index_t& loop_index,
    nvfuser_index_t buffer_size) {
  constexpr nvfuser_index_t offset =
      loop_offset < 0 ? (loop_offset + number_of_stage) : loop_offset;

  // Rewind back at number_of_stage-1, otherwise increment by 1.
  nvfuser_index_t increment =
      (loop_index % number_of_stage) == (number_of_stage - 1 - offset)
      ? buffer_size * (-number_of_stage + 1)
      : buffer_size;
  buffer_offset += (int)increment;
}

// Reset smem space to zero
// TODO: try cp.async.ignore-source ?
template <typename dtype, int len>
DEVICE_INLINE void smemReset(SmemAddress smem_addr) {
  constexpr int byte_size = sizeof(dtype) * len;

  static_assert(
      byte_size == 4 || byte_size == 8 || byte_size == 16,
      "cp_async : unsupported byte size");

  switch (byte_size) {
    case 4:
      asm volatile(
          "{\n"
          "st.shared.u32 [%0], {%1};\n"
          "}\n"
          :
          : "r"(smem_addr), "r"(0));
      break;
    case 8:
      asm volatile(
          "{\n"
          "st.shared.v2.u32 [%0], {%1, %2};\n"
          "}\n"
          :
          : "r"(smem_addr), "r"(0), "r"(0));
      break;
    case 16:
      asm volatile(
          "{\n"
          "st.shared.v4.u32 [%0], {%1, %2, %3, %4};\n"
          "}\n"
          :
          : "r"(smem_addr), "r"(0), "r"(0), "r"(0), "r"(0));
      break;
  }
}

#undef DEVICE_INLINE
